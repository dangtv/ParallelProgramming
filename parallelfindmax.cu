
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 512 //


__global__ void findmax(float * input, float * output, int len) {
	__shared__ float partialMax[BLOCK_SIZE];
	
	unsigned int t = threadIdx.x;
	unsigned int start = blockIdx.x * blockDim.x;
	
	partialMax[t] = (start + t < len) ? input[start + t] : 0;
	__syncthreads();


	for(unsigned int stride = blockDim.x/4; stride >= 1; stride >>= 1) {
		if(t < stride){
			partialMax[t] = (partialMax[t] < partialMax[t + stride]) ? partialMax[t + stride]:partialMax[t];
			partialMax[t] = (partialMax[t] < partialMax[t + 2*stride]) ? partialMax[t + 2*stride]:partialMax[t];
			partialMax[t] = (partialMax[t] < partialMax[t + 3*stride]) ? partialMax[t + 3*stride]:partialMax[t];
		}
		__syncthreads();
	}
	
	if(t == 0) {
		output[blockIdx.x + t] = partialMax[t];
	}

}

int main(int argc, char ** argv) {
 
    int ii;
    float * hostInput; // The input 1D array
    float * hostOutput; // The output array
    float * deviceInput;
    float * deviceOutput;
    int numInputElements = 4099; // number of elements in the input array
    int numOutputElements; // number of elements in the output array


    hostInput = (float*) malloc(numInputElements * sizeof(float));
    for(int i = 0; i < numInputElements; i++) {
        hostInput[i] = (i-2000)*(i-2000);
    }

    numOutputElements = numInputElements / (BLOCK_SIZE);
    if (numInputElements % (BLOCK_SIZE)) {
        numOutputElements++;
    }
    hostOutput = (float*) malloc(numOutputElements * sizeof(float));


	hipMalloc((void **) &deviceInput, numInputElements * sizeof(float));
	hipMalloc((void **) &deviceOutput, numOutputElements * sizeof(float));
	

	
	hipMemcpy(deviceInput, hostInput, numInputElements * sizeof(float), hipMemcpyHostToDevice);


	
	dim3 DimGrid((numInputElements - 1)/BLOCK_SIZE + 1, 1, 1);
	dim3 DimBlock(BLOCK_SIZE, 1, 1);


	findmax<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, numInputElements);
	
    hipDeviceSynchronize();

	hipMemcpy(hostOutput, deviceOutput, numOutputElements * sizeof(float), hipMemcpyDeviceToHost);
	


    for (ii = 1; ii < numOutputElements; ii++) {
        hostOutput[0] = (hostOutput[0]<hostOutput[ii]) ? hostOutput[ii]:hostOutput[0];
    }
    printf("%f\n", hostOutput[0]);

	hipFree(deviceInput);
	hipFree(deviceOutput);

    free(hostInput);
    free(hostOutput);    
  
    return 0;
}
