// MP 4 Reduction
// Given a list (lst) of length n
// Output its sum = lst[0] + lst[1] + ... + lst[n-1];
// Due Tuesday, January 15, 2013 at 11:59 p.m. PST

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 512 //@@ You can change this


__global__ void total(float * input, float * output, int len) {
	__shared__ float partialMax[2 * BLOCK_SIZE];
	
	unsigned int t = threadIdx.x;
	unsigned int start = 2 * blockIdx.x * blockDim.x;
	
	partialMax[t] = (t < len) ? input[start + t] : 0;
	partialMax[blockDim.x + t] = ((blockDim.x + t) < len) ? input[start + blockDim.x + t] : 0;
	
	//@@ Load a segment of the input vector into shared memory
	
	for(unsigned int stride = blockDim.x; stride >= 1; stride >>= 1) {
		__syncthreads();
		if(t < stride)
			partialMax[t] = (partialMax[t] < partialMax[t + stride]) ? partialMax[t + stride]:partialMax[t];
	}
	
	if(t == 0) {
		output[blockIdx.x + t] = partialMax[t];
	}
	//@@ Traverse the reduction tree
    //@@ Write the computed sum of the block to the output vector at the 
    //@@ correct index    //@@ Load a segment of the input vector into shared memory
    //@@ Traverse the reduction tree
    //@@ Write the computed sum of the block to the output vector at the 
    //@@ correct index
}

int main(int argc, char ** argv) {
 
    int ii;
    float * hostInput; // The input 1D list
    float * hostOutput; // The output list
    float * deviceInput;
    float * deviceOutput;
    int numInputElements = 2048; // number of elements in the input list
    int numOutputElements; // number of elements in the output list


    hostInput = (float*) malloc(numInputElements * sizeof(float));
    for(int i = 0; i < numInputElements; i++) {
        hostInput[i] = i;
    }

    numOutputElements = numInputElements / (BLOCK_SIZE<<1);
    if (numInputElements % (BLOCK_SIZE<<1)) {
        numOutputElements++;
    }
    hostOutput = (float*) malloc(numOutputElements * sizeof(float));

    //@@ Allocate GPU memory here
	hipMalloc((void **) &deviceInput, numInputElements * sizeof(float));
	hipMalloc((void **) &deviceOutput, numOutputElements * sizeof(float));
	
    //@@ Copy memory to the GPU here
	
	hipMemcpy(deviceInput, hostInput, numInputElements * sizeof(float), hipMemcpyHostToDevice);

    //@@ Initialize the grid and block dimensions here
	
	dim3 DimGrid((numInputElements - 1)/BLOCK_SIZE + 1, 1, 1);
	dim3 DimBlock(BLOCK_SIZE, 1, 1);

    //@@ Launch the GPU Kernel here
	total<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, numInputElements);
	
    hipDeviceSynchronize();

    //@@ Copy the GPU memory back to the CPU here
	hipMemcpy(hostOutput, deviceOutput, numOutputElements * sizeof(float), hipMemcpyDeviceToHost);
	

    /********************************************************************
     * Reduce output vector on the host
     * NOTE: One could also perform the reduction of the output vector
     * recursively and support any size input. For simplicity, we do not
     * require that for this lab.
     ********************************************************************/
    for (ii = 1; ii < numOutputElements; ii++) {
        hostOutput[0] = (hostOutput[0]<hostOutput[ii]) ? hostOutput[ii]:hostOutput[0];
    }
    printf("%f\n", hostOutput[0]);

    //@@ Free the GPU memory here
	hipFree(deviceInput);
	hipFree(deviceOutput);

    free(hostInput);
    free(hostOutput);    
  
    return 0;
}
