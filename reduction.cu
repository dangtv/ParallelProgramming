// MP 4 Reduction
// Given a list (lst) of length n
// Output its sum = lst[0] + lst[1] + ... + lst[n-1];
// Due Tuesday, January 15, 2013 at 11:59 p.m. PST

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 512 //@@ You can change this


__global__ void findmax(float * input, float * output, int len) {
	__shared__ float partialMax[BLOCK_SIZE];
	
	unsigned int t = threadIdx.x;
	unsigned int start = blockIdx.x * blockDim.x;
	
	partialMax[t] = (t < len) ? input[start + t] : 0;
	__syncthreads();
	//@@ Load a segment of the input vector into shared memory

	for(unsigned int stride = blockDim.x/2; stride >= 1; stride >>= 1) {
		if(t < stride)
			partialMax[t] = (partialMax[t] < partialMax[t + stride]) ? partialMax[t + stride]:partialMax[t];
		__syncthreads();
	}
	
	if(t == 0) {
		output[blockIdx.x + t] = partialMax[t];
	}
	//@@ Traverse the reduction tree
    //@@ Write the computed sum of the block to the output vector at the 
    //@@ correct index    //@@ Load a segment of the input vector into shared memory
    //@@ Traverse the reduction tree
    //@@ Write the computed sum of the block to the output vector at the 
    //@@ correct index
}

int main(int argc, char ** argv) {
 
    int ii;
    float * hostInput; // The input 1D list
    float * hostOutput; // The output list
    float * deviceInput;
    float * deviceOutput;
    int numInputElements = 40960; // number of elements in the input list
    int numOutputElements; // number of elements in the output list


    hostInput = (float*) malloc(numInputElements * sizeof(float));
    for(int i = 0; i < numInputElements; i++) {
        hostInput[i] = i;
    }

    numOutputElements = numInputElements / (BLOCK_SIZE);
    if (numInputElements % (BLOCK_SIZE)) {
        numOutputElements++;
    }
    hostOutput = (float*) malloc(numOutputElements * sizeof(float));

    //@@ Allocate GPU memory here
	hipMalloc((void **) &deviceInput, numInputElements * sizeof(float));
	hipMalloc((void **) &deviceOutput, numOutputElements * sizeof(float));
	
    //@@ Copy memory to the GPU here
	
	hipMemcpy(deviceInput, hostInput, numInputElements * sizeof(float), hipMemcpyHostToDevice);

    //@@ Initialize the grid and block dimensions here
	
	dim3 DimGrid((numInputElements - 1)/BLOCK_SIZE + 1, 1, 1);
	dim3 DimBlock(BLOCK_SIZE, 1, 1);

    //@@ Launch the GPU Kernel here
	findmax<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, numInputElements);
	
    hipDeviceSynchronize();

    //@@ Copy the GPU memory back to the CPU here
	hipMemcpy(hostOutput, deviceOutput, numOutputElements * sizeof(float), hipMemcpyDeviceToHost);
	

    /********************************************************************
     * Reduce output vector on the host
     * NOTE: One could also perform the reduction of the output vector
     * recursively and support any size input. For simplicity, we do not
     * require that for this lab.
     ********************************************************************/
    for (ii = 1; ii < numOutputElements; ii++) {
        hostOutput[0] = (hostOutput[0]<hostOutput[ii]) ? hostOutput[ii]:hostOutput[0];
    }
    printf("%f\n", hostOutput[0]);

    //@@ Free the GPU memory here
	hipFree(deviceInput);
	hipFree(deviceOutput);

    free(hostInput);
    free(hostOutput);    
  
    return 0;
}
