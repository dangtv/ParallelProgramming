
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>


#define N 10000
#define BlX 100

__global__ void Cong(int *a, int *b, int *c) {
	int i;
	i = blockIdx.x*blockDim.x + threadIdx.x;
	*(c + i) = *(a + i) + *(b + i);
}

int main()
{
	int *Ah, *Bh, *Ch, *Ad, *Bd, *Cd;
	int size = N * sizeof(int);
	Ah = (int*)malloc(size);
	Bh = (int*)malloc(size);
	Ch = (int*)malloc(size);

	hipMalloc((void**)&Ad, size);
	hipMalloc((void**)&Bd, size);
	hipMalloc((void**)&Cd, size);

	for (int i = 0; i < N; i++)
	{
		*(Ah + i) = i;
		*(Bh + i) = 2 * i;
	}
	hipMemcpy(Ad, Ah, size, hipMemcpyHostToDevice);
	hipMemcpy(Bd, Bh, size, hipMemcpyHostToDevice);

	dim3 dimBlock(BlX, 1, 1);
	dim3 dimGrid(N / BlX, 1, 1);

	Cong<<<dimGrid, dimBlock>>>(Ad, Bd, Cd);
	hipMemcpy(Ch, Cd, size, hipMemcpyDeviceToHost);
	for (int i = 0; i < 20; i++)
	{
		printf("%d\n", *(Ch + i));
	}

	free(Ah); free(Bh); free(Ch);
	hipFree(Ad); hipFree(Bd); hipFree(Cd);
	return 0;
}